#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#define TOTAL_COL 20480


__global__ void copy_128B(
    const float* __restrict__ d_in,
    float* __restrict__ d_out,
    size_t total_columns
){
    int row = blockIdx.x;
    int col = blockIdx.y * blockDim.x + threadIdx.x;
    int idx = row * total_columns + col;
    float out = __ldg(d_in + idx);
    if (out > TOTAL_COL * 64) d_out[idx] = out;
}


__global__ void copy_32B(
    const float* __restrict__ d_in,
    float* __restrict__ d_out,
    size_t total_columns
){
    int row = threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * total_columns + col;
    float out = __ldg(d_in + idx);
    if (out > TOTAL_COL * 64) d_out[idx] = out;
}

int main(void){
    float *h_in = new float[64 * TOTAL_COL];
    float *h_out = new float[64 * TOTAL_COL];

    for (int i = 0; i < 64 * TOTAL_COL; i ++){
        h_in[i] = float(i);
    }

    float *d_in, *d_out;

    hipMalloc(&d_in, 64 * TOTAL_COL * sizeof(float));
    hipMalloc(&d_out, 64 * TOTAL_COL * sizeof(float));
    hipMemcpy(d_in, h_in, 64 * TOTAL_COL * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out, h_out, 64 * TOTAL_COL * sizeof(float), hipMemcpyHostToDevice);

    

    // copy_128B<<<grid_dim, block_dim>>>(d_in, d_out, TOTAL_COL);
    // copy_128B<<<grid_dim, block_dim>>>(d_in, d_out, TOTAL_COL);
    // copy_128B<<<grid_dim, block_dim>>>(d_in, d_out, TOTAL_COL);
    // copy_128B<<<grid_dim, block_dim>>>(d_in, d_out, TOTAL_COL);

    hipProfilerStart();
    dim3 grid_dim(64, TOTAL_COL / 512, 1);
    dim3 block_dim(512, 1, 1);
    copy_128B<<<grid_dim, block_dim>>>(d_in, d_out, TOTAL_COL);
    
    // dim3 grid_dim(TOTAL_COL / 8, 1, 1);
    // dim3 block_dim(8, 64, 1);

    // copy_32B<<<grid_dim, block_dim>>>(d_in, d_out, TOTAL_COL);

    hipProfilerStop();

    hipMemcpy(h_out, d_out, 64 * TOTAL_COL * sizeof(float), hipMemcpyDeviceToHost);

    int errors = 0;
    for (int j=0; j < 64 * TOTAL_COL; j++){
        if (abs(h_out[j] - h_in[j]) > 0.001){
            errors ++;
        }
    }

    if (errors > 0) {
        printf( "COPY does not agree with SEQUENTIAL! %d errors!\n",errors);
    }
    else {
        printf("Results verified: they agree.\n");
    }

    hipFree(d_in);
    hipFree(d_out);
    delete h_in;
    delete h_out;
}
