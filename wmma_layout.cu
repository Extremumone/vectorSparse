
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <stdio.h>
#include <mma.h>
using namespace nvcuda;

__device__ void print_val_h(int threadid, half value, int reg){
    printf("index %.0f, tid: %d, register %d\n", float(value), threadid, reg);
}

__global__ void dummy_wmma(float* d_out){
    // The shared memory that holds the indices
    __shared__ half indices[512];

    // Step 1: write the indices to the shared memory
    for (int i=threadIdx.x; i < 512; i+=blockDim.x){
        indices[i] = half(i);
    }

    __syncthreads();

    // Step 2: declare the fragments
    wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::col_major> a_frag;
    wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, 8, 32, 16, float> c_frag;

    // Step 3: load the fragments
    wmma::load_matrix_sync(a_frag, indices, 8);

    // #pragma unroll
    // for (int i = 0; i < 16; i++){
    //     print_val_h(threadIdx.x, a_frag.x[i], i);
    // }

    wmma::load_matrix_sync(b_frag, indices, 32);

    #pragma unroll
    for (int i = 0; i < 16; i++){
        print_val_h(threadIdx.x, b_frag.x[i], i);
    }

    wmma::fill_fragment(c_frag, 0.0f);

    // Step 4: do GEMM
    wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

    // Step 5: stoore the matrix
    wmma::store_matrix_sync(d_out, c_frag, 32, wmma::mem_row_major);
}


int main(void){
    float* d_out;
    hipMalloc(&d_out, 256 * sizeof(float));

    dummy_wmma<<<1, 32>>>(d_out);

    float* h_out = new float[256];

    hipMemcpy(h_out, d_out, 256 * sizeof(float), hipMemcpyDeviceToHost);
    for (int i=0; i < 8; i++){
        for (int j=0; j < 32; j++){
            printf("%.0f ", h_out[i * 32 + j]);
        }
        printf("\n");
    }
}